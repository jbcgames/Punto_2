#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gputimer.h"
#include "cputimer.h"

#include <iostream>
#include <iomanip>
#include <cmath>
#include <random>
#include <vector>
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#define _IA64_ 1
static void CheckCudaErrorAux(const char*, unsigned, const char*, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__, __LINE__, #value, value)

// Declaración de kernels
__global__ void dotProductKernel_v1(float* A, float* B, float* acum, unsigned size) {
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        atomicAdd(acum, A[idx] * B[idx]);
    }
}

__global__ void dotProductKernel_v2(float* A, float* B, float* acum, unsigned size) {
    __shared__ float temp[1024];
    int gIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int lIdx = threadIdx.x;
    temp[lIdx] = A[gIdx] * B[gIdx];
    __syncthreads();
    if (lIdx == 0) {
        for (int k = 1; k < blockDim.x; k++) {
            temp[0] += temp[k];
        }
        atomicAdd(acum, temp[0]);
    }
}

__global__ void dotProductKernel_v3(float* A, float* B, float* acum, unsigned size) {
    __shared__ float temp[1024];
    int gIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int lIdx = threadIdx.x;
    if (gIdx < size) {
        temp[lIdx] = A[gIdx] * B[gIdx];
    } else {
        temp[lIdx] = 0;
    }

    int k = blockDim.x / 2;
    while (k > 0) {
        if (lIdx < k) {
            temp[lIdx] += temp[lIdx + k];
        }
        k /= 2;
        __syncthreads();
    }
    if (lIdx == 0) {
        atomicAdd(acum, temp[0]);
    }
}

// Declaración de funciones
float gpuDotProduct(float* A, float* B, unsigned size, unsigned numT, int kernelVersion) ;
float cpuDotProduct(float* A, float* B, unsigned size);
void initFloatVec(float* data, unsigned size);
void gpuPrintProperties(unsigned GpuID);
static void CheckCudaErrorAux(const char* file, unsigned line, const char* statement, hipError_t err);
float gputimeT, gputimeC;

int main(void) {
    unsigned deviceID = 0;
    static const int MAX_THREAD = 1024;
    std::vector<int> workSizes;
	
    // Generar tamaños de trabajo: desde 2^10 hasta 2^30
    for (int i = 10; i <= 30; ++i) {
        workSizes.push_back(1 << i);
    }

    std::cout << std::setw(20) << "WORK_SIZE " 
              << std::setw(15) << "CPU_TIME" 
              << std::setw(20) << "GPU_TIME_KERNEL_V1" 
              << std::setw(20) << "GPU_TIME_KERNEL_V2" 
              << std::setw(20) << "GPU_TIME_KERNEL_V3"
              << std::setw(20) << "GPU_TIME_TOTAL"  
              << std::endl;

    for (int WORK_SIZE : workSizes) {
        float* A = new float[WORK_SIZE];
        float* B = new float[WORK_SIZE];
        float acumCpu, acumGpu;

        initFloatVec(A, WORK_SIZE);
        initFloatVec(B, WORK_SIZE);

        CpuTimer cpuTimer1;
        CUDA_CHECK_RETURN(hipSetDevice(deviceID));

        // Medir el tiempo en CPU
        cpuTimer1.Start();
        acumCpu = cpuDotProduct(A, B, WORK_SIZE);
        cpuTimer1.Stop();

        // Medir el tiempo en GPU con diferentes kernels
        for (int kernelVersion = 1; kernelVersion <= 3; ++kernelVersion) {
            acumGpu = gpuDotProduct(A, B, WORK_SIZE, MAX_THREAD, kernelVersion);
            std::cout << std::setw(15) << WORK_SIZE 
                      << std::setw(20) << 1000 * cpuTimer1.Elapsed().count() 
                      << std::setw(20) << ((kernelVersion == 1) ? gputimeT : 0) 
                      << std::setw(20) << ((kernelVersion == 2) ? gputimeT : 0)
                      << std::setw(20) << ((kernelVersion == 3) ? gputimeT : 0) 
                      << std::setw(20) << gputimeC
                      << std::endl;
        }

        delete[] A;
        delete[] B;
    }

    return 0;
}


float gpuDotProduct(float* A, float* B, unsigned size, unsigned numT, int kernelVersion) {
    float* gpuA;
    float* gpuB;
    float* gpuAcum;
    float acum;
    GpuTimer gpuTimer1, gpuTimer2;
    
    gpuTimer1.Start();
    CUDA_CHECK_RETURN(hipMalloc((void**)&gpuA, sizeof(float) * size));
    CUDA_CHECK_RETURN(hipMalloc((void**)&gpuB, sizeof(float) * size));
    CUDA_CHECK_RETURN(hipMalloc((void**)&gpuAcum, sizeof(float)));
    CUDA_CHECK_RETURN(hipMemcpy(gpuA, A, sizeof(float) * size, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(gpuB, B, sizeof(float) * size, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemset(gpuAcum, 0, sizeof(float)));

    static const int BLOCK_SIZE = numT;
    const int blockCount = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    gpuTimer2.Start();
    switch (kernelVersion) {
        case 1:
            dotProductKernel_v1<<<blockCount, BLOCK_SIZE>>>(gpuA, gpuB, gpuAcum, size);
            break;
        case 2:
            dotProductKernel_v2<<<blockCount, BLOCK_SIZE>>>(gpuA, gpuB, gpuAcum, size);
            break;
        case 3:
            dotProductKernel_v3<<<blockCount, BLOCK_SIZE>>>(gpuA, gpuB, gpuAcum, size);
            break;
        default:
            std::cerr << "Invalid kernel version" << std::endl;
            exit(EXIT_FAILURE);
    }
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    gpuTimer2.Stop();

    CUDA_CHECK_RETURN(hipMemcpy(&acum, gpuAcum, sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipFree(gpuA));
    CUDA_CHECK_RETURN(hipFree(gpuB));
    CUDA_CHECK_RETURN(hipFree(gpuAcum));
    gpuTimer1.Stop();

    gputimeT = gpuTimer2.Elapsed();
    gputimeC = gpuTimer1.Elapsed();

    return acum;
}

float cpuDotProduct(float* A, float* B, unsigned size) {
    float acum = 0;
    for (unsigned i = 0; i < size; ++i) {
        acum += A[i] * B[i];
    }
    return acum;
}

void initFloatVec(float* data, unsigned size) {
    std::random_device rd;
    for (unsigned i = 0; i < size; ++i) {
        data[i] = static_cast<float>(rd()) / static_cast<float>(UINT32_MAX);
    }
}

static void CheckCudaErrorAux(const char* file, unsigned line, const char* statement, hipError_t err) {
    if (err == hipSuccess) return;
    std::cerr << statement << " returned " << hipGetErrorString(err) << "(" << err << ") at " << file << ":" << line << std::endl;
    exit(1);
}
